#include<stdio.h>
#include<time.h>
#include<malloc.h>
#include<hip/hip_runtime.h>

#define N 32
#define n_threads 4
#define n_blocks 8

void __global__ kernelTong2Mang(int *a, int *b, int *c) {
    int index;
    index = blockIdx.x * blockDim.x + threadIdx.x;
    *(c + index) = *(a + index) + *(b + index);
}

int main(int argc, char *argv[]) {
    // HOST CODE
    int *a_cpu, *b_cpu, *c_cpu, i;
    a_cpu = (int*) malloc(N * sizeof(int));
    b_cpu = (int*) malloc(N * sizeof(int));
    c_cpu = (int*) malloc(N * sizeof(int));

    for (i = 0; i < N; i ++) {
        *(a_cpu + i) = i;
        *(b_cpu + i) = i;
    }
    
    // Khai bao bien tren GPU
    int *a_gpu, *b_gpu, *c_gpu;
    hipMalloc((void**) &a_gpu, N * sizeof(int));
    hipMalloc((void**) &b_gpu, N * sizeof(int));
    hipMalloc((void**) &c_gpu, N * sizeof(int));
    
    // Copy input CPU -> GPU
    hipMemcpy(a_gpu, a_cpu, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(b_gpu, b_cpu, N * sizeof(int), hipMemcpyHostToDevice);
    
    // Define structure: thread, block
    dim3 dimBlock(n_threads);
    dim3 dimGrid(n_blocks);
    
    // Invoke kernel
    kernelTong2Mang<<<dimGrid, dimBlock>>>(a_gpu, b_gpu, c_gpu);
    
    // Copy output GPU -> CPU
    hipMemcpy(c_cpu, c_gpu, N * sizeof(int), hipMemcpyDeviceToHost);
    
    // Giai phong bo nho GPU
    hipFree(a_gpu);
    hipFree(b_gpu);
    hipFree(c_gpu);
    
    for (i = 0; i < N; i ++) {
        printf("%d\t", *(c_cpu + i));
    }
    
    return 0;
}

